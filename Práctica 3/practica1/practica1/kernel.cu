
#include "hip/hip_runtime.h"


#include <iostream>

__global__ void suma_vectores(float *a,float *b,float *c,int n) {
	int idx_ = blockIdx.x * blockDim.x + threadIdx.x;
//	c[idx_] = a[idx_] + b[idx_];

	// Ejercicios adicionales
	if(idx_ < n) {
		for (int i = idx_; i < n; i = blockDim.x * gridDim.x + i) {
			c[i] = a[i] + b[i];
		}
	}
}

int main(void) {
	const int kNumElement = 256001;
	const int kNumBytes = sizeof(float) * kNumElement;

	hipSetDevice(0);

	// decleramos nuestros vectores y rellenarlos
	float* h_a_ = (float *)malloc(kNumBytes); // todo lo que tenga que ver con procesar, se pone h, de host
	float* h_b_ = (float *)malloc(kNumBytes);
	float* h_c_ = (float *)malloc(kNumBytes);

	// comprobamos si hay error
	if(h_a_ == NULL  ||  h_b_ == NULL  ||  h_c_ == NULL) {
		std::cerr << "Error al reservar memoria\n";
		getchar();
		exit(-1);
	}

	// rellenamos vectores
	for(int i=0; i<kNumElement; i++) {
		h_a_[i] = rand() / (float)RAND_MAX;
		h_b_[i] = rand() / (float)RAND_MAX;
	}

	// tenemos que pasar los datos de la cpu a la gpu
	float * d_a_ = NULL;
	float * d_b_ = NULL;
	float * d_c_ = NULL;

	// tenemos que decirlo cual es el puntero sobre el que inicializamos y el tamanyo de bytes
	hipMalloc((void **)&d_a_, kNumBytes);
	hipMalloc((void **)&d_b_, kNumBytes);
	hipMalloc((void **)&d_c_, kNumBytes);

	// transferimos los arrays a la GPU desde la CPU
	hipMemcpy(d_a_, h_a_, kNumBytes, hipMemcpyHostToDevice) ;
	hipMemcpy(d_b_, h_b_, kNumBytes, hipMemcpyHostToDevice);

	// cuantos hilos por bloques queremos tener

	int threads_per_block_ = 256;
	int blocks_per_grid_;
	
	if(kNumElement%threads_per_block_ == 0) {
		blocks_per_grid_ = kNumElement/threads_per_block_;
	} else {
		blocks_per_grid_ = (kNumElement/threads_per_block_)+1;
	}
	
	// modificamos el tipo de datos que le gusta mas a cuda
	dim3 tpb_(threads_per_block_,1 ,1);
	dim3 bpg_(blocks_per_grid_, 1, 1);

	suma_vectores<<< bpg_, tpb_ >>>(d_a_, d_b_, d_c_,kNumElement);

	//para controlar errores
	hipError_t err_ = hipGetLastError();
	if(err_ != hipSuccess)
	{
		std::cerr << "Error " << hipGetErrorString(err_) << '\n';
		getchar();
		exit(-1);
	}

	hipMemcpy(h_c_, d_c_, kNumBytes, hipMemcpyDeviceToHost);

	for(int i=0; i<kNumElement; i++) {
		if(fabs(h_a_[i] + h_b_[i] - h_c_[i]) > 1e-5) {
			std::cerr << "Error en la posicion " << i << "\n";
			getchar();
			exit(-1);
		}
	}

	free(h_a_);
	free(h_b_);
	free(h_c_);

	hipFree(d_a_);
	hipFree(d_b_);
	hipFree(d_c_);

	std::cout << "Optimo\n";
	getchar();
	exit(0);

}
